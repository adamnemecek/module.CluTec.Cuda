#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
// project:   CluTec.Cuda.ImgProc
// file:      Statistics.DevFromMean.cu
//
// summary:   statistics. development from mean class
//
//            Copyright (c) 2019 by Christian Perwass.
//
//            This file is part of the CluTecLib library.
//
//            The CluTecLib library is free software: you can redistribute it and / or modify
//            it under the terms of the GNU Lesser General Public License as published by
//            the Free Software Foundation, either version 3 of the License, or
//            (at your option) any later version.
//
//            The CluTecLib library is distributed in the hope that it will be useful,
//            but WITHOUT ANY WARRANTY; without even the implied warranty of
//            MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//            GNU Lesser General Public License for more details.
//
//            You should have received a copy of the GNU Lesser General Public License
//            along with the CluTecLib library.
//            If not, see <http://www.gnu.org/licenses/>.
//
////////////////////////////////////////////////////////////////////////////////////////////////////

//#define CLU_DEBUG_KERNEL

#include "hip/hip_runtime.h"

#include "Statistics.DevFromMean.h"
#include "CluTec.Types1/Pixel.h"
#include "CluTec.Math/Conversion.h"

#include "CluTec.Cuda.Base/Kernel.ArrayCache.h"
#include "CluTec.Cuda.Base/Kernel.Debug.h"
#include "Kernel.Algo.ArraySum.h"

namespace Clu
{
	namespace Cuda
	{
		namespace Statistics
		{
			namespace DevFromMean
			{
				namespace Kernel
				{
					using namespace Clu::Cuda::Kernel;

					template<int t_iPatchWidth, int t_iPatchHeight
						, int t_iPatchCountY_Pow2
						, int t_iWarpsPerBlockX, int t_iWarpsPerBlockY>
						struct Constants
					{
						// Warps per block
						static const int WarpsPerBlockX = t_iWarpsPerBlockX;
						static const int WarpsPerBlockY = t_iWarpsPerBlockY;

						// Thread per warp
						static const int ThreadsPerWarp = 32;

						static const int ThreadsPerBlockX = WarpsPerBlockX * ThreadsPerWarp;

						// the width of a base patch has to be a full number of words
						static const int BasePatchSizeX = t_iPatchWidth;
						static const int BasePatchSizeY = t_iPatchHeight;
						static const int BasePatchElementCount = BasePatchSizeX * BasePatchSizeY;

						using AlgoSum = Clu::Cuda::Kernel::AlgoArraySum<ThreadsPerBlockX, t_iPatchCountY_Pow2,
							BasePatchSizeX, BasePatchSizeY>;

						static const int BasePatchCountX = AlgoSum::PatchCountX;
						static const int BasePatchCountY = AlgoSum::PatchCountY;

						static const int TestBlockSizeX = AlgoSum::DataArraySizeX;
						static const int TestBlockSizeY = AlgoSum::DataArraySizeY;

						static const int SumCacheSizeX = AlgoSum::SumCacheSizeX;
						static const int SumCacheSizeY = AlgoSum::SumCacheSizeY;

						static const int ResultCountPerThread = AlgoSum::ColGroupSizeX;


#define PRINT(theVar) printf(#theVar ": %d\n", theVar)

						__device__ static void PrintValues()
						{
							printf("Block Idx: %d, %d\n", blockIdx.x, blockIdx.y);
							PRINT(ThreadsPerBlockX);
							PRINT(BasePatchSizeX);
							PRINT(BasePatchSizeY);
							PRINT(BasePatchCountX);
							PRINT(BasePatchCountY);
							PRINT(TestBlockSizeX);
							PRINT(TestBlockSizeY);
							PRINT(ResultCountPerThread);
							//PRINT();
							printf("\n");

							__syncthreads();
						}
#undef PRINT
					};

					////////////////////////////////////////////////////////////////////////////////////////////////////
					/// <summary>	The algorithm parameters. </summary>
					///
					/// <value>	. </value>
					////////////////////////////////////////////////////////////////////////////////////////////////////

					__constant__ _SParameter c_xPars;

					////////////////////////////////////////////////////////////////////////////////////////////////////
					/// <summary>	Transform image. </summary>
					///
					/// <typeparam name="TPixel">	Type of the pixel. </typeparam>
					/// <param name="xImageTrg">	The image trg. </param>
					/// <param name="xImageSrc">	The image source. </param>
					////////////////////////////////////////////////////////////////////////////////////////////////////

					template<typename TPixel, EConfig t_eConfig>
					__global__ void Filter(Clu::Cuda::_CDeviceSurface xImageTrg, Clu::Cuda::_CDeviceSurface xImageSrc)
					{
						using Config = SConfig<t_eConfig>;
						using Const = Kernel::Constants<Config::PatchSizeX, Config::PatchSizeY, Config::PatchCountY_Pow2, Config::WarpsPerBlockX, Config::WarpsPerBlockY>;

						using AlgoSum = typename Const::AlgoSum;
						using TData = typename TPixel::TData;
						using TSum = int;// unsigned short;

						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////

						// Get position of thread in left image
						const int iBlockX = blockIdx.x * Const::BasePatchCountX;
						const int iBlockY = blockIdx.y * Const::BasePatchCountY;

						if (!xImageSrc.Format().IsRectInside(iBlockX, iBlockY, Const::TestBlockSizeX, Const::TestBlockSizeY))
						{
							return;
						}

						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						Debug::Run([]()
						{
							if (Debug::IsThreadAndBlock(0, 0, 0, 0))
							{
								Const::PrintValues();
							}
						});

						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						__shared__ Clu::Cuda::Kernel::CArrayCache<TSum
							, Const::SumCacheSizeX, Const::SumCacheSizeY
							, Const::WarpsPerBlockX, Const::WarpsPerBlockY, 8, 1>
							xCacheSumY;


						TSum piDevMean[Const::ResultCountPerThread];

						// //////////////////////////////////////////////////////////////////////////////////////////////
						// //////////////////////////////////////////////////////////////////////////////////////////////
						Const::AlgoSum::SumCache(xCacheSumY,
							// //////////////////////////////////////////////////////////////////////////////////////////////
							[&xImageSrc, &iBlockX, &iBlockY](int iIdxX, int iIdxY)
						{
							return (TSum)xImageSrc.Read2D<TData>(iBlockX + iIdxX, iBlockY + iIdxY);
						},
							// //////////////////////////////////////////////////////////////////////////////////////////////
							[&piDevMean, &xImageSrc, &iBlockX, &iBlockY](TSum& iValue, int iResultIdx, int iIdxX, int iIdxY)
						{
							TSum iMean = iValue / Const::BasePatchElementCount;

							TSum iCenter = (TSum)xImageSrc.Read2D<TData>(
								iBlockX + iIdxX + Const::BasePatchSizeX / 2 + iResultIdx
								, iBlockY + iIdxY + Const::BasePatchSizeY / 2);

							piDevMean[iResultIdx] = (iCenter - iMean) /** TSum(2)*/ + (TSum(Clu::NumericLimits<TData>::Max()) >> 1);
						});
						// //////////////////////////////////////////////////////////////////////////////////////////////
						// //////////////////////////////////////////////////////////////////////////////////////////////


						for (int iResultIdx = 0; iResultIdx < Const::ResultCountPerThread; ++iResultIdx)
						{
							TData uValue = (TData)min(max(piDevMean[iResultIdx], 0), (TSum)Clu::NumericLimits<TData>::Max());

							xImageTrg.Write2D(uValue
								, AlgoSum::RowSumBaseIdxX() + iBlockX + Const::BasePatchSizeX / 2 + iResultIdx
								, AlgoSum::RowSumBaseIdxY() + iBlockY + Const::BasePatchSizeY / 2);
						}
					}

					////////////////////////////////////////////////////////////////////////////////////////////////////
					/// <summary>	Filter with no border. </summary>
					///
					/// <typeparam name="TPixel">   	Type of the pixel. </typeparam>
					/// <typeparam name="t_eConfig">	Type of the configuration. </typeparam>
					/// <param name="xImageTrg">	The image trg. </param>
					/// <param name="xImageSrc">	The image source. </param>
					////////////////////////////////////////////////////////////////////////////////////////////////////

					template<typename TPixel, EConfig t_eConfig>
					__global__ void FilterNoBorder(Clu::Cuda::_CDeviceSurface xImageTrg, Clu::Cuda::_CDeviceSurface xImageSrc)
					{
						using Config = SConfig<t_eConfig>;
						using Const = Kernel::Constants<Config::PatchSizeX, Config::PatchSizeY, Config::PatchCountY_Pow2, Config::WarpsPerBlockX, Config::WarpsPerBlockY>;

						using AlgoSum = typename Const::AlgoSum;
						using TData = typename TPixel::TData;
						using TSum = int;// unsigned short;

						 // ////////////////////////////////////////////////////////////////////////////////////////////////
						 // ////////////////////////////////////////////////////////////////////////////////////////////////
						 // ////////////////////////////////////////////////////////////////////////////////////////////////
						 
						 // Get position of thread in left image
						const int iCtrX = blockIdx.x * Const::BasePatchCountX;
						const int iCtrY = blockIdx.y * Const::BasePatchCountY;

						const int iW = xImageSrc.Format().iWidth;
						const int iH = xImageSrc.Format().iHeight;

						int iPatchX = iCtrX - Const::BasePatchSizeX / 2;
						int iPatchY = iCtrY - Const::BasePatchSizeY / 2;

						
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						Debug::Run([]()
						{
							if (Debug::IsThreadAndBlock(0, 0, 0, 0))
							{
								Const::PrintValues();
							}
						});

						//if (!Debug::IsBlock(52, 20) && !Debug::IsBlock(53, 20))
						//{
						//	return;
						//}

						//if (Debug::IsThread(0, 0))
						//{
						//	printf("Block: %d, %d\nImage Size: %d, %d\nCenter: %d, %d\nPatch: %d, %d\n", iBlockX, iBlockY, iW, iH, iCtrX, iCtrY, iPatchX, iPatchY);
						//}

						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						__shared__ Clu::Cuda::Kernel::CArrayCache<TSum
							, Const::SumCacheSizeX, Const::SumCacheSizeY
							, Const::WarpsPerBlockX, Const::WarpsPerBlockY, 8, 1>
							xCacheSumY;


						// //////////////////////////////////////////////////////////////////////////////////////////////
						// //////////////////////////////////////////////////////////////////////////////////////////////
						Const::AlgoSum::SumCache(xCacheSumY,
						// //////////////////////////////////////////////////////////////////////////////////////////////
							[&xImageSrc, &iPatchX, &iPatchY, &iW, &iH](int iIdxX, int iIdxY)
						{
							int iX = iPatchX + iIdxX;
							int iY = iPatchY + iIdxY;

							iX += int(iX < 0) * Const::BasePatchSizeX;
							iY += int(iY < 0) * Const::BasePatchSizeY;
							iX -= int(iX >= iW) * Const::BasePatchSizeX;
							iY -= int(iY >= iH) * Const::BasePatchSizeY;

							if (iX < iW && iY < iH)
							{
								return (TSum)xImageSrc.Read2D<TData>(iX, iY);
							}
							else
							{
								return TSum(0);
							}
						},
						// //////////////////////////////////////////////////////////////////////////////////////////////
							[&](TSum& iValue, int iResultIdx, int iIdxX, int iIdxY)
						{
							const int iX = iCtrX + iIdxX + iResultIdx;
							const int iY = iCtrY + iIdxY;

							if (iX >= iW || iY >= iH)
							{
								return;
							}

							const float fMean = float(iValue) / float(Const::BasePatchElementCount);

							const float fCenter = (float)xImageSrc.Read2D<TData>(iX, iY);

							// Gamma adjustment of resultant image
							float fResult = (fCenter - fMean)  / float(Clu::NumericLimits<TData>::Max());
							const float fHalfMax = float(Clu::NumericLimits<TData>::Max() >> 1);

							//fResult = c_xPars.fScale
							//	* (fResult < 0.0f ? -1.0f : 1.0f)
							//	* powf(abs(fResult), c_xPars.fGamma)
							//	* fHalfMax
							//	+ fHalfMax;

							if (abs(fResult) <= c_xPars.fCtrX)
							{
								fResult = (fResult < 0.0f ? -1.0f : 1.0f)
									* c_xPars.fCtrY
									* powf(abs(fResult) / c_xPars.fCtrX, c_xPars.fGamma1);
							}
							else
							{
								fResult = (fResult < 0.0f ? -1.0f : 1.0f)
										* (1.0f - (1.0f - c_xPars.fCtrY)
												* powf((1.0f - abs(fResult)) / (1.0f - c_xPars.fCtrX), c_xPars.fGamma2));
							}

							fResult *= fHalfMax;
							fResult += fHalfMax;

							const TSum iResult = TSum(floor(fResult + 0.5f));
							const TData uValue = (TData)Clu::Clamp(iResult, TSum(0), (TSum)Clu::NumericLimits<TData>::Max());

							xImageTrg.Write2D(uValue, iX, iY);
						});
						// //////////////////////////////////////////////////////////////////////////////////////////////
						// //////////////////////////////////////////////////////////////////////////////////////////////

					}

				}

				template<EConfig t_eConfig>
				void CDriver::_DoConfigure(const Clu::Cuda::CDevice& xDevice, const Clu::SImageFormat& xFormat)
				{
					using Config = SConfig<t_eConfig>;
					using Const = Kernel::Constants<Config::PatchSizeX, Config::PatchSizeY, Config::PatchCountY_Pow2, Config::WarpsPerBlockX, Config::WarpsPerBlockY>;

					unsigned nOffsetLeft = 0; 
					unsigned nOffsetRight = 0;
					unsigned nOffsetTop = 0; 
					unsigned nOffsetBottom = 0; 

					EvalThreadConfigBlockSize(xDevice, xFormat
						, Const::BasePatchCountX, Const::BasePatchCountY
						, nOffsetLeft, nOffsetRight, nOffsetTop, nOffsetBottom
						, Config::WarpsPerBlockX, Config::WarpsPerBlockY
						, Config::NumberOfRegisters
						, false // do process partial blocks
					);
				}

				////////////////////////////////////////////////////////////////////////////////////////////////////
				/// <summary>	Configures. </summary>
				///
				/// <param name="xConfig">	[in,out] The configuration. </param>
				/// <param name="xDevice">	The device. </param>
				/// <param name="xFormat">	Describes the format to use. </param>
				////////////////////////////////////////////////////////////////////////////////////////////////////

#define _CLU_DO_CONFIG(theId) \
			case theId: \
				_DoConfigure<theId>(xDevice, xFormat); \
				break

				void CDriver::Configure(const Clu::Cuda::CDevice& xDevice, const Clu::SImageFormat& xFormat,
					const SParameter& xPars)
				{
					m_xPars = xPars;

					switch (m_xPars.eConfig)
					{
						_CLU_DO_CONFIG(EConfig::Patch_16x16);
						_CLU_DO_CONFIG(EConfig::Patch_11x11);
						_CLU_DO_CONFIG(EConfig::Patch_9x9);
						_CLU_DO_CONFIG(EConfig::Patch_7x7);
						_CLU_DO_CONFIG(EConfig::Patch_5x5);
						_CLU_DO_CONFIG(EConfig::Patch_3x3);

					default:
						throw CLU_EXCEPTION("Invalid algorithm configuration.");
					}

				}
#undef _CLU_DO_CONFIG

#define _CLU_DO_PROCESS(theId) \
			case theId: \
				Kernel::FilterNoBorder<TPixel, theId> \
					CLU_KERNEL_CONFIG() \
					(xImageOut, xImageIn); \
				break

				template<typename TPixel>
				void CDriver::_DoProcess(Clu::Cuda::_CDeviceSurface& xImageOut
					, const Clu::Cuda::_CDeviceSurface& xImageIn)
				{
					switch (m_xPars.eConfig)
					{
						_CLU_DO_PROCESS(EConfig::Patch_16x16);
						_CLU_DO_PROCESS(EConfig::Patch_11x11);
						_CLU_DO_PROCESS(EConfig::Patch_9x9);
						_CLU_DO_PROCESS(EConfig::Patch_7x7);
						_CLU_DO_PROCESS(EConfig::Patch_5x5);
						_CLU_DO_PROCESS(EConfig::Patch_3x3);

					default:
						throw CLU_EXCEPTION("Invalid algorithm configuration.");
					}

				}
#undef _CLU_DO_PROCESS

				////////////////////////////////////////////////////////////////////////////////////////////////////
				/// <summary>	Process this object. </summary>
				///
				/// <param name="dimBlocksInGrid">   	The dim blocks in grid. </param>
				/// <param name="dimThreadsPerBlock">	The dim threads per block. </param>
				/// <param name="xImageDisp">		 	[in,out] The image disp. </param>
				/// <param name="xImageL">			 	The image l. </param>
				/// <param name="xImageR">			 	The image r. </param>
				/// <param name="iOffset">			 	Zero-based index of the offset. </param>
				/// <param name="iDispRange">		 	Zero-based index of the disp range. </param>
				/// <param name="fSadThresh">		 	The sad thresh. </param>
				/// <param name="iMinDeltaThresh">   	Zero-based index of the minimum delta thresh. </param>
				////////////////////////////////////////////////////////////////////////////////////////////////////

				void CDriver::Process(Clu::Cuda::_CDeviceSurface& xImageOut
					, const Clu::Cuda::_CDeviceSurface& xImageIn)
				{
					Clu::Cuda::MemCpyToSymbol(Kernel::c_xPars, &m_xPars, 1, 0, Clu::Cuda::ECopyType::HostToDevice);

					if (!xImageOut.IsEqualType(xImageIn.Format()))
					{
						throw CLU_EXCEPTION("Given output image has different type than input image");
					}

					if (xImageIn.IsOfType<Clu::TPixel_Lum_UInt8>()
						&& xImageOut.IsOfType<Clu::TPixel_Lum_UInt8>())
					{
						_DoProcess<Clu::TPixel_Lum_UInt8>(xImageOut, xImageIn);
					}
					else
					{
						throw CLU_EXCEPTION("Pixel types of given images not supported");
					}
				}




			} // Mean
		} // Statistics
	} // Cuda
} // Clu

