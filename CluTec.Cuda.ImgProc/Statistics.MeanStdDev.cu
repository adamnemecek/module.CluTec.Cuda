#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
// project:   CluTec.Cuda.ImgProc
// file:      Statistics.MeanStdDev.cu
//
// summary:   statistics. mean standard development class
//
//            Copyright (c) 2019 by Christian Perwass.
//
//            This file is part of the CluTecLib library.
//
//            The CluTecLib library is free software: you can redistribute it and / or modify
//            it under the terms of the GNU Lesser General Public License as published by
//            the Free Software Foundation, either version 3 of the License, or
//            (at your option) any later version.
//
//            The CluTecLib library is distributed in the hope that it will be useful,
//            but WITHOUT ANY WARRANTY; without even the implied warranty of
//            MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//            GNU Lesser General Public License for more details.
//
//            You should have received a copy of the GNU Lesser General Public License
//            along with the CluTecLib library.
//            If not, see <http://www.gnu.org/licenses/>.
//
////////////////////////////////////////////////////////////////////////////////////////////////////

#include "hip/hip_runtime.h"

#include "Statistics.MeanStdDev.h"
#include "CluTec.Types1/Pixel.h"
#include "CluTec.Math/Conversion.h"

#include "CluTec.Cuda.Base/Kernel.ArrayCache.h"
#include "CluTec.Cuda.Base/Kernel.Debug.h"
#include "Kernel.Algo.ArraySum.h"

namespace Clu
{
	namespace Cuda
	{
		namespace Statistics
		{
			namespace MeanStdDev
			{
				namespace Kernel
				{
					using namespace Clu::Cuda::Kernel;

					template<int t_iPatchWidth, int t_iPatchHeight
						, int t_iPatchCountY_Pow2
						, int t_iWarpsPerBlockX, int t_iWarpsPerBlockY>
					struct Constants
					{
						// Warps per block
						static const int WarpsPerBlockX = t_iWarpsPerBlockX;
						static const int WarpsPerBlockY = t_iWarpsPerBlockY;

						// Thread per warp
						static const int ThreadsPerWarp = 32;

						static const int ThreadsPerBlockX = WarpsPerBlockX * ThreadsPerWarp;

						// the width of a base patch has to be a full number of words
						static const int BasePatchSizeX = t_iPatchWidth;
						static const int BasePatchSizeY = t_iPatchHeight;
						static const int BasePatchElementCount = BasePatchSizeX * BasePatchSizeY;

						using AlgoSum = Clu::Cuda::Kernel::AlgoArraySum<ThreadsPerBlockX, t_iPatchCountY_Pow2,
							BasePatchSizeX, BasePatchSizeY>;

						static const int BasePatchCountX = AlgoSum::PatchCountX;
						static const int BasePatchCountY = AlgoSum::PatchCountY;

						static const int TestBlockSizeX = AlgoSum::DataArraySizeX + 1;
						static const int TestBlockSizeY = AlgoSum::DataArraySizeY;

						static const int SumCacheSizeX = AlgoSum::SumCacheSizeX;
						static const int SumCacheSizeY = AlgoSum::SumCacheSizeY;

						static const int ResultCountPerThread = AlgoSum::ColGroupSizeX;


#define PRINT(theVar) printf(#theVar ": %d\n", theVar)

						__device__ static void PrintValues()
						{
							printf("Block Idx: %d, %d\n", blockIdx.x, blockIdx.y);
							PRINT(ThreadsPerBlockX);
							PRINT(BasePatchSizeX);
							PRINT(BasePatchSizeY);
							PRINT(BasePatchCountX);
							PRINT(BasePatchCountY);
							PRINT(TestBlockSizeX);
							PRINT(TestBlockSizeY);
							PRINT(ResultCountPerThread);
							//PRINT();
							printf("\n");

							__syncthreads();
						}
#undef PRINT
					};

					////////////////////////////////////////////////////////////////////////////////////////////////////
					/// <summary>	The algorithm parameters. </summary>
					///
					/// <value>	. </value>
					////////////////////////////////////////////////////////////////////////////////////////////////////

					__constant__ _SParameter c_xPars;



					__device__ void ResultToPixel(Clu::TPixel_LumA_UInt8& pixResult, float fMean, float fStdDev)
					{
						using TPixelTrg = Clu::TPixel_LumA_UInt8;
						using TData = typename TPixelTrg::TData;
						using TProd = int;

						pixResult.r() = (TData)min(TProd(floor(fMean + 0.5f)), (TProd)Clu::NumericLimits<TData>::Max());
						pixResult.a() = (TData)min(TProd(floor(fStdDev + 0.5f)), (TProd)Clu::NumericLimits<TData>::Max());
					}

					__device__ void ResultToPixel(Clu::TPixel_LumA_Single& pixResult, float fMean, float fStdDev)
					{
						pixResult.r() = fMean;
						pixResult.a() = fStdDev;
					}

					////////////////////////////////////////////////////////////////////////////////////////////////////
					/// <summary>	Filter with no border. </summary>
					///
					/// <typeparam name="TPixel">   	Type of the pixel. </typeparam>
					/// <typeparam name="t_eConfig">	Type of the configuration. </typeparam>
					/// <param name="xImageTrg">	The image trg. </param>
					/// <param name="xImageSrc">	The image source. </param>
					////////////////////////////////////////////////////////////////////////////////////////////////////

					template<typename TPixelTrg, typename TPixelSrc, EConfig t_eConfig>
					__global__ void FilterNoBorder(Clu::Cuda::_CDeviceSurface xImageTrg, Clu::Cuda::_CDeviceSurface xImageSrc)
					{
						using Config = SConfig<t_eConfig>;
						using Const = Kernel::Constants<Config::PatchSizeX, Config::PatchSizeY, Config::PatchCountY_Pow2, Config::WarpsPerBlockX, Config::WarpsPerBlockY>;

						using TElement = typename Clu::Cuda::SPixelTypeInfo<TPixelSrc>::TElement;

						using AlgoSum = typename Const::AlgoSum;
						using TData = typename TPixelSrc::TData;
						using TSum = int;//short;
						using TProd = int;

						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////

						// Get position of thread in left image
						const int iCtrX = blockIdx.x * Const::BasePatchCountX;
						const int iCtrY = blockIdx.y * Const::BasePatchCountY;

						const int iW = xImageSrc.Format().iWidth;
						const int iH = xImageSrc.Format().iHeight;

						int iPatchX = iCtrX - Const::BasePatchSizeX / 2;
						int iPatchY = iCtrY - Const::BasePatchSizeY / 2;

						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						Debug::Run([]()
						{
							if (Debug::IsThreadAndBlock(0, 0, 0, 0))
							{
								Const::PrintValues();
							}
						});

						//if (!Debug::IsBlock(52, 20) && !Debug::IsBlock(53, 20))
						//{
						//	return;
						//}

						//if (Debug::IsThread(0, 0))
						//{
						//	printf("Block: %d, %d\nImage Size: %d, %d\nCenter: %d, %d\nPatch: %d, %d\n", iBlockX, iBlockY, iW, iH, iCtrX, iCtrY, iPatchX, iPatchY);
						//}
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						__shared__ Clu::Cuda::Kernel::CArrayCache<TElement
							, Const::TestBlockSizeX, Const::TestBlockSizeY
							, Const::WarpsPerBlockX, Const::WarpsPerBlockY, 8, 1>
							xCachePatch;


						__shared__ Clu::Cuda::Kernel::CArrayCache<TSum
							, Const::SumCacheSizeX, Const::SumCacheSizeY
							, Const::WarpsPerBlockX, Const::WarpsPerBlockY, 8, 1>
							xCacheSumY;


						// //////////////////////////////////////////////////////////////////////////////////////////////
						// ////////////////////////////////////////////////////////////////////////////////////////////////
						int iOffsetX, iOffsetY;
						iOffsetX = min(iPatchX, max(0, iPatchX + Const::TestBlockSizeX - iW));
						iOffsetY = min(iPatchY, max(0, iPatchY + Const::TestBlockSizeY - iH));

						if (xImageSrc.IsRectInside(iPatchX - iOffsetX, iPatchY - iOffsetY, Const::TestBlockSizeX, Const::TestBlockSizeY))
						{
							xCachePatch.ReadFromSurf<Const::TestBlockSizeX, Const::TestBlockSizeY>(xImageSrc, iPatchX - iOffsetX, iPatchY - iOffsetY);
						}
						__syncthreads();


						// //////////////////////////////////////////////////////////////////////////////////////////////
						// //////////////////////////////////////////////////////////////////////////////////////////////
						TSum piMean[Const::ResultCountPerThread];
						TSum piMean2[Const::ResultCountPerThread];

						// //////////////////////////////////////////////////////////////////////////////////////////////
						// //////////////////////////////////////////////////////////////////////////////////////////////
						Const::AlgoSum::SumCache(xCacheSumY,
							// //////////////////////////////////////////////////////////////////////////////////////////////
							[&xCachePatch, &iPatchX, &iPatchY, &iOffsetX, &iOffsetY, &iW, &iH](int iIdxX, int iIdxY)
						{
							int iX = iPatchX + iIdxX;
							int iY = iPatchY + iIdxY;

							iX += int(iX < 0) * Const::BasePatchSizeX;
							iY += int(iY < 0) * Const::BasePatchSizeY;
							iX -= int(iX >= iW) * Const::BasePatchSizeX;
							iY -= int(iY >= iH) * Const::BasePatchSizeY;

							if (iX < iW && iY < iH)
							{
								return (TSum)xCachePatch.At(iX - iPatchX + iOffsetX, iY - iPatchY + iOffsetY).x;
							}
							else
							{
								return TSum(0);
							}
						},
							// //////////////////////////////////////////////////////////////////////////////////////////////
							[&piMean, &xImageSrc, &iCtrX, &iCtrY, &iW, &iH](TSum& iValue, int iResultIdx, int iIdxX, int iIdxY)
						{
							const int iX = iCtrX + iIdxX + iResultIdx;
							const int iY = iCtrY + iIdxY;

							if (iX < iW && iY < iH)
							{
								piMean[iResultIdx] = iValue;
							}
							else
							{
								piMean[iResultIdx] = TSum(0);
							}
						});

						// //////////////////////////////////////////////////////////////////////////////////////////////
						// //////////////////////////////////////////////////////////////////////////////////////////////
						Const::AlgoSum::SumCache(xCacheSumY,
							// //////////////////////////////////////////////////////////////////////////////////////////////
							[&xCachePatch, &iPatchX, &iPatchY, &iOffsetX, &iOffsetY, &iW, &iH](int iIdxX, int iIdxY)
						{
							int iX = iPatchX + iIdxX;
							int iY = iPatchY + iIdxY;

							iX += int(iX < 0) * Const::BasePatchSizeX;
							iY += int(iY < 0) * Const::BasePatchSizeY;
							iX -= int(iX >= iW) * Const::BasePatchSizeX;
							iY -= int(iY >= iH) * Const::BasePatchSizeY;

							if (iX < iW && iY < iH)
							{
								TProd iValue = (TProd)xCachePatch.At(iX - iPatchX + iOffsetX, iY - iPatchY + iOffsetY).x;
								return (TSum) (iValue * iValue);
							}
							else
							{
								return TSum(0);
							}
						},
							// //////////////////////////////////////////////////////////////////////////////////////////////
							[&piMean2, &xImageSrc, &iCtrX, &iCtrY, &iW, &iH](TSum& iValue, int iResultIdx, int iIdxX, int iIdxY)
						{
							const int iX = iCtrX + iIdxX + iResultIdx;
							const int iY = iCtrY + iIdxY;

							if (iX < iW && iY < iH)
							{
								piMean2[iResultIdx] = iValue;
							}
							else
							{
								piMean2[iResultIdx] = TSum(0);
							}
						});

						// //////////////////////////////////////////////////////////////////////////////////////////////
						// //////////////////////////////////////////////////////////////////////////////////////////////
						const float fBasePatchElementCount = float(Const::BasePatchElementCount);

						for (int iResultIdx = 0; iResultIdx < Const::ResultCountPerThread; ++iResultIdx)
						{
							const int iX = AlgoSum::RowSumBaseIdxX() + iCtrX + iResultIdx;
							const int iY = AlgoSum::RowSumBaseIdxY() + iCtrY;

							if (iX < iW && iY < iH)
							{
								const TProd iMean = (TProd)piMean[iResultIdx];
								const TProd iMean2 = (TProd)piMean2[iResultIdx];

								//uMean2 <<= 8;

								const float fMean2 = float(iMean2) / fBasePatchElementCount;
								const float fMean = float(iMean) / fBasePatchElementCount;

								const float fStdDev = (c_xPars.fScale * sqrtf((fMean2 - fMean * fMean)));
								TPixelTrg pixResult;

								ResultToPixel(pixResult, fMean, fStdDev);

								xImageTrg.WritePixel2D(pixResult, iX, iY);
							}


						}
					}

				}

				template<EConfig t_eConfig>
				void CDriver::_DoConfigure(const Clu::Cuda::CDevice& xDevice, const Clu::SImageFormat& xFormat)
				{
					using Config = SConfig<t_eConfig>;
					using Const = Kernel::Constants<Config::PatchSizeX, Config::PatchSizeY, Config::PatchCountY_Pow2, Config::WarpsPerBlockX, Config::WarpsPerBlockY>;

					unsigned nOffsetLeft	= 0; //Config::PatchSizeX / 2;
					unsigned nOffsetRight	= 0; //Config::PatchSizeX / 2;
					unsigned nOffsetTop		= 0; //Config::PatchSizeY / 2;
					unsigned nOffsetBottom	= 0; //Config::PatchSizeY / 2;

					EvalThreadConfigBlockSize(xDevice, xFormat
						, Const::BasePatchCountX, Const::BasePatchCountY
						, nOffsetLeft, nOffsetRight, nOffsetTop, nOffsetBottom
						, Config::WarpsPerBlockX, Config::WarpsPerBlockY
						, Config::NumberOfRegisters
						, false // do process partial blocks
						);
				}

				////////////////////////////////////////////////////////////////////////////////////////////////////
				/// <summary>	Configures. </summary>
				///
				/// <param name="xConfig">	[in,out] The configuration. </param>
				/// <param name="xDevice">	The device. </param>
				/// <param name="xFormat">	Describes the format to use. </param>
				////////////////////////////////////////////////////////////////////////////////////////////////////

#define _CLU_DO_CONFIG(theId) \
			case theId: \
				_DoConfigure<theId>(xDevice, xFormat); \
				break

				void CDriver::Configure(const Clu::Cuda::CDevice& xDevice, const Clu::SImageFormat& xFormat,
					const SParameter& xPars)
				{
					m_xPars = xPars;

					switch (m_xPars.eConfig)
					{
						_CLU_DO_CONFIG(EConfig::Patch_16x16);
						_CLU_DO_CONFIG(EConfig::Patch_11x11);
						_CLU_DO_CONFIG(EConfig::Patch_9x9);
						_CLU_DO_CONFIG(EConfig::Patch_7x7);
						_CLU_DO_CONFIG(EConfig::Patch_5x5);
						_CLU_DO_CONFIG(EConfig::Patch_3x3);

					default:
						throw CLU_EXCEPTION("Invalid algorithm configuration.");
					}

				}
#undef _CLU_DO_CONFIG

#define _CLU_DO_PROCESS(theId) \
			case theId: \
				Kernel::FilterNoBorder<TPixelTrg, TPixelSrc, theId> \
					CLU_KERNEL_CONFIG() \
					(xImageOut, xImageIn); \
				break

				template<typename TPixelTrg, typename TPixelSrc>
				void CDriver::_DoProcess(Clu::Cuda::_CDeviceSurface& xImageOut
					, const Clu::Cuda::_CDeviceSurface& xImageIn)
				{
					switch (m_xPars.eConfig)
					{
						_CLU_DO_PROCESS(EConfig::Patch_16x16);
						_CLU_DO_PROCESS(EConfig::Patch_11x11);
						_CLU_DO_PROCESS(EConfig::Patch_9x9);
						_CLU_DO_PROCESS(EConfig::Patch_7x7);
						_CLU_DO_PROCESS(EConfig::Patch_5x5);
						_CLU_DO_PROCESS(EConfig::Patch_3x3);

					default:
						throw CLU_EXCEPTION("Invalid algorithm configuration.");
					}

				}
#undef _CLU_DO_PROCESS

				////////////////////////////////////////////////////////////////////////////////////////////////////
				/// <summary>	Process this object. </summary>
				///
				/// <param name="dimBlocksInGrid">   	The dim blocks in grid. </param>
				/// <param name="dimThreadsPerBlock">	The dim threads per block. </param>
				/// <param name="xImageDisp">		 	[in,out] The image disp. </param>
				/// <param name="xImageL">			 	The image l. </param>
				/// <param name="xImageR">			 	The image r. </param>
				/// <param name="iOffset">			 	Zero-based index of the offset. </param>
				/// <param name="iDispRange">		 	Zero-based index of the disp range. </param>
				/// <param name="fSadThresh">		 	The sad thresh. </param>
				/// <param name="iMinDeltaThresh">   	Zero-based index of the minimum delta thresh. </param>
				////////////////////////////////////////////////////////////////////////////////////////////////////

				void CDriver::Process(Clu::Cuda::_CDeviceSurface& xImageOut
					, const Clu::Cuda::_CDeviceSurface& xImageIn)
				{
					Clu::Cuda::MemCpyToSymbol(Kernel::c_xPars, &m_xPars, 1, 0, Clu::Cuda::ECopyType::HostToDevice);

					if (!xImageOut.IsEqualSize(xImageIn.Format()))
					{
						throw CLU_EXCEPTION("Given output image has different dimensions than input image");
					}

					if (xImageIn.IsOfType<Clu::TPixel_Lum_UInt8>()
						&& xImageOut.IsOfType<Clu::TPixel_LumA_UInt8>())
					{
						_DoProcess<Clu::TPixel_LumA_UInt8, Clu::TPixel_Lum_UInt8>(xImageOut, xImageIn);
					}
					else if (xImageIn.IsOfType<Clu::TPixel_Lum_UInt8>()
						&& xImageOut.IsOfType<Clu::TPixel_LumA_Single>())
					{
						_DoProcess<Clu::TPixel_LumA_Single, Clu::TPixel_Lum_UInt8>(xImageOut, xImageIn);
					}
					else
					{
						throw CLU_EXCEPTION("Pixel types of given images not supported");
					}
				}


				// /////////////////////////////////////////////////////////////////////////
				// Snippet: calculate variance
				// unsigned char pucIsBelowVarThresh[Const::BasePatchCountSplitX];
				//
				// {
				//
				//	// Evaluate mean value of base patches.
				//	Const::AlgoSum::SumCache(xCacheSumY,
				//		[&xCacheBasePatch](int iIdxX, int iIdxY)
				//	{
				//		auto pixVal = xCacheBasePatch.At(iIdxX, iIdxY);
				//		return TSum(pixVal.x);
				//	},
				//		[&pxMinDisp](TSum& iValue, int iRelX, int iIdxX, int iIdxY)
				//	{
				//		pxMinDisp[iRelX].x = iValue / Const::BasePatchElementCount;
				//	}
				//	);
				//
				//	// Evaluate mean deviation from mean value
				//	Const::AlgoSum::SumCache(xCacheSumY,
				//		[&xCacheBasePatch](int iIdxX, int iIdxY)
				//	{
				//		auto pixVal = xCacheBasePatch.At(iIdxX, iIdxY);
				//		return TSum(pixVal.x >> 4) * TSum(pixVal.x >> 4);
				//	},
				//		[&pxMinDisp](TSum& iValue, int iRelX, int iIdxX, int iIdxY)
				//	{
				//		pxMinDisp[iRelX].y = iValue / Const::BasePatchElementCount;
				//	}
				//	);
				//
				//	int iIsBelowVarThresh = 1;
				//	for (int iRelX = 0; iRelX < Const::BasePatchCountSplitX; ++iRelX)
				//	{
				//		float fMean = float(pxMinDisp[iRelX].x);
				//		float fMean2 = float(pxMinDisp[iRelX].y);
				//		float fVar = fMean2 - fMean * fMean;
				//
				//		//if (IsBlock(40, 40))
				//		//{
				//		//	printf("%g\n", fVar);
				//		//}
				//
				//		int iIsBelow = int(fVar < 5.0f);
				//		pucIsBelowVarThresh[iRelX] = (unsigned char)iIsBelow;
				//		iIsBelowVarThresh *= iIsBelow;
				//
				//		//static const float fVarMax = 5000.0f;
				//		//fVar /= fVarMax;
				//
				//		////TDispComp ucValue = Clu::NormFloatTo<TDispComp>(fVar);
				//		//TDisp pixDisp = Clu::Cuda::NormFloatToColor<TPixelDisp>(fVar);
				//
				//		//surfDisp.WritePixel2D<TPixelDisp>(pixDisp //make_uchar4(ucValue, ucValue, ucValue, 255)
				//		//	, AlgoSum::BaseIdxX() + iDispX + iRelX
				//		//	, AlgoSum::BaseIdxY() + iDispY);
				//	}
				//
				//	if (__syncthreads_and(iIsBelowVarThresh) > 0)
				//	{
				//		return;
				//	}
				//
				// }




			} // Mean
		} // Statistics
	} // Cuda
} // Clu

